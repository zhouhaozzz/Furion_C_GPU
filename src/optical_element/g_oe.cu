#include "hip/hip_runtime.h"
#include "G_Oe.h"
#include "g_beam.h"
#include <chrono>

using namespace Furion_NS;

G_Oe::G_Oe(G_Beam* beam_in, real_t ds, real_t di, real_t chi, real_t theta, No_Surfe* surface, Grating* grating)
    : beam_in(beam_in), grating(grating), surface(surface), theta(theta), chi(chi)//, beam_out(beam_in), Cff(0), theta2(0)
{
    cout << "G_Oe ��ʼ��" << endl;
}

G_Oe::~G_Oe()
{
    delete[] X_, Y_, PHI, PSI, Phase;
    delete[] L1, M1, N1, X1, Y1, Z1, X2, Y2, Z2, cos_Alpha;
    cout << "~G_Oe������" << endl;

}

__global__ void Furion_NS::g_oe_cuda(real_t* X_, real_t* Y_, real_t* Phase, real_t* X3, real_t* Y3, real_t* Z3, real_t* L3, real_t* M3, real_t* N3, real_t* T, real_t* Z2, real_t lambda, real_t m, real_t n0, real_t b, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    real_t T1;

    if (i < n)
    {
        T1 = -Z3[i] / N3[i];
        X_[i] = X3[i] + T1 * L3[i];
        Y_[i] = Y3[i] + T1 * M3[i];
        Phase[i] = (T[i] + T1) / lambda * 2 * Pi - n0 * m * 2 * Pi * Z2[i] - 0.5 * m * b * n0 * 2 * Pi * (Z2[i] * Z2[i]);
    }

    __syncthreads();
}

void G_Oe::g_oe_GPU(real_t* X_, real_t* Y_, real_t* Phase, real_t* X3, real_t* Y3, real_t* Z3, real_t* L3, real_t* M3, real_t* N3, real_t* T, real_t* Z2, real_t lambda, real_t m, real_t n0, real_t b)
{
    int n = Furion::n;
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    //auto start = std::chrono::high_resolution_clock::now();
    //srand((unsigned)time(NULL));

    //real_t* d_X_, * d_Y_, * d_Phase;
    //real_t* d_X3, * d_Y3, * d_Z3;
    //real_t* d_L3, * d_M3, * d_N3, * d_T, * d_Z2;
    //hipMalloc((void**)&d_X3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Y3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Z3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_L3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_M3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_N3, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_X_, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Y_, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_T, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Z2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Phase, Furion::n * sizeof(real_t));

    //hipMemcpy(d_L3, L3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_M3, M3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_N3, N3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_X3, X3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_Y3, Y3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_Z3, Z3, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_Z2, Z2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_T, T, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);

    //auto start1 = std::chrono::high_resolution_clock::now();
    //srand((unsigned)time(NULL));

    Furion_NS::g_oe_cuda << <blocksPerGrid, threadsPerBlock >> > (X_, Y_, Phase, X3, Y3, Z3, L3, M3, N3, T, Z2, lambda, m, n0, b, n);

    //auto end1 = std::chrono::high_resolution_clock::now();
    //auto duration1 = std::chrono::duration_cast<std::chrono::microseconds>(end1 - start1);
    //std::cout << "GPU Execution time: " << duration1.count() / 1e6 << " seconds" << std::endl;

    //hipMemcpy(this->X_, d_X_, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->Y_, d_Y_, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->Phase, d_Phase, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    //hipFree(d_X_);
    //hipFree(d_Y_);
    //hipFree(d_Phase);
    //hipFree(d_X3);
    //hipFree(d_Y3);
    //hipFree(d_Z3);
    //hipFree(d_L3);
    //hipFree(d_N3);
    //hipFree(d_M3);
    //hipFree(d_Z2);
    //hipFree(d_T);
    //auto end = std::chrono::high_resolution_clock::now();
    //auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    //std::cout << "Total Execution time: " << duration.count() / 1e6 << " seconds" << std::endl;
}

void G_Oe::reflect(G_Beam* beam_in, real_t ds, real_t di, real_t chi, real_t theta)
{
    //real_t* L = new real_t[Furion::n];
    //real_t* M = new real_t[Furion::n];
    //real_t* N = new real_t[Furion::n];
    //real_t* T = new real_t[Furion::n];
    //real_t* T1 = new real_t[Furion::n];
    //real_t* Nx = new real_t[Furion::n];
    //real_t* Ny = new real_t[Furion::n];
    //real_t* Nz = new real_t[Furion::n];
    //real_t* hslope = new real_t[Furion::n];
    //real_t* L2 = new real_t[Furion::n];
    //real_t* M2 = new real_t[Furion::n];
    //real_t* N2 = new real_t[Furion::n];
    //real_t* X3 = new real_t[Furion::n];
    //real_t* Y3 = new real_t[Furion::n];
    //real_t* Z3 = new real_t[Furion::n];
    //real_t* L3 = new real_t[Furion::n];
    //real_t* M3 = new real_t[Furion::n];
    //real_t* N3 = new real_t[Furion::n];

    real_t* d_L, * d_M, * d_N;
    hipMalloc((void**)&d_L, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N, Furion::n * sizeof(real_t));
    f_a_v.Furion_angle_vector(beam_in->phi, beam_in->psi, d_L, d_M, d_N);               //[phi,psi]-&gt; [L,M,N] angles are converted to unit vectors

    real_t* d_X1, * d_Y1, * d_Z1;
    real_t* d_L1, * d_M1, * d_N1;
    real_t* d_XX, * d_YY;
    hipMalloc((void**)&d_X1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Z1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_L1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_XX, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_YY, Furion::n * sizeof(real_t));
    hipMemcpy(d_XX, beam_in->XX, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_YY, beam_in->YY, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    source_to_oe(d_X1, d_Y1, d_Z1, d_L1, d_M1, d_N1, d_XX, d_YY, ds, d_L, d_M, d_N);           //From light source coordinate system to optical component coordinate system
    
    hipFree(d_XX);
    hipFree(d_YY);
    hipFree(d_L);
    hipFree(d_M);
    hipFree(d_N);
    hipMemcpy(this->X1, d_X1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Y1, d_Y1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Z1, d_Z1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->L1, d_L1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->M1, d_M1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->N1, d_N1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    
    real_t* d_X2, * d_Y2, * d_Z2, * d_T;
    hipMalloc((void**)&d_X2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Z2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_T, Furion::n * sizeof(real_t));
    intersection(d_T, d_X2, d_Y2, d_Z2, d_X1, d_Y1, d_Z1, d_L1, d_M1, d_N1);                                      //The intersection of light and optical components
    hipMemcpy(this->X2, d_X2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Y2, d_Y2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Z2, d_Z2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipFree(d_X1);
    hipFree(d_Y1);
    hipFree(d_Z1);

    real_t* d_Nx, * d_Ny, * d_Nz;
    hipMalloc((void**)&d_Nx, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Ny, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Nz, Furion::n * sizeof(real_t));
    normal(d_X2, d_Y2, d_Z2, d_Nx, d_Ny, d_Nz);                                   //mormal Find reflection vector

    real_t* d_hslope;
    hipMalloc((void**)&d_hslope, Furion::n * sizeof(real_t));
    h_slope(d_X2, d_Y2, d_Z2, d_L1, d_N1, d_hslope);     //Calculate the surface slope error    Find the slope of the corresponding position

    this->theta2 = Pi / 2 - asin(sin(Pi / 2 - theta) - grating->n0 * grating->m * grating->lambda_G);
    this->Cff = cos(Pi / 2 - this->theta2) / cos(Pi / 2 - this->theta);

    real_t* d_cos_Alpha;
    real_t* d_L2, * d_M2, * d_N2;
    hipMalloc((void**)&d_cos_Alpha, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_L2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N2, Furion::n * sizeof(real_t));
    f_r_v.Furion_reflect_Vector(d_cos_Alpha, d_L2, d_M2, d_N2, d_L1, d_M1, d_N1, d_Nx, d_Ny, d_Nz, grating->lambda_G, grating->m, grating->n0, grating->b, d_Z2, d_hslope, this->Cff);
    
    hipMemcpy(cos_Alpha, d_cos_Alpha, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->M1, d_M1, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipFree(d_L1);
    hipFree(d_M1);
    hipFree(d_N1);
    hipFree(d_Nx);
    hipFree(d_Ny);
    hipFree(d_Nz);
    hipFree(d_cos_Alpha);
    hipFree(d_hslope);

    real_t* d_X3, * d_Y3, * d_Z3;
    real_t* d_L3, * d_M3, * d_N3;
    hipMalloc((void**)&d_X3, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y3, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Z3, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_L3, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M3, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N3, Furion::n * sizeof(real_t));
    oe_to_image(d_X3, d_Y3, d_Z3, d_L3, d_M3, d_N3, d_X2, d_Y2, d_Z2, di, d_L2, d_M2, d_N2);
    hipMemcpy(this->X2, d_X2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Z2, d_Z2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    hipFree(d_X2);
    hipFree(d_Y2);
    //hipFree(d_Z2);
    hipFree(d_L2);
    hipFree(d_M2);
    hipFree(d_N2);

    real_t* d_X_, * d_Y_, * d_Phase;
    hipMalloc((void**)&d_X_, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y_, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Phase, Furion::n * sizeof(real_t));
    g_oe_GPU(d_X_, d_Y_, d_Phase, d_X3, d_Y3, d_Z3, d_L3, d_M3, d_N3, d_T, d_Z2, grating->lambda_G, grating->m, grating->n0, grating->b);
    

    
    hipMemcpy(this->X_, d_X_, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Y_, d_Y_, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipMemcpy(this->Phase, d_Phase, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    hipFree(d_Z2);
    hipFree(d_X3);
    hipFree(d_Y3);
    hipFree(d_Z3);
    hipFree(d_N3);
    hipFree(d_T);
    hipFree(d_X_);
    hipFree(d_Y_);
    hipFree(d_Phase);

    f_v_a.Furion_vector_angle(this->PHI, this->PSI, d_L3, d_M3);
    hipFree(d_L3);
    hipFree(d_M3);

    //real_t* L = new real_t[Furion::n];
    //real_t* M = new real_t[Furion::n];
    //real_t* N = new real_t[Furion::n];
    //hipMemcpy(L, d_L3, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(M, d_M3, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(N, d_N3, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //ofstream fileout("data.dat");
    //fileout << std::fixed;
    //fileout << std::setprecision(15);
    //for (int i = 0; i < Furion::n; i++)
    //{
    //    fileout << this->X_[i] << " ";
    //}
    //fileout << "\n";

    //for (int i = 0; i < Furion::n; i++)
    //{
    //    fileout << this->Y_[i] << " ";
    //}
    //fileout << "\n";

    //for (int i = 0; i < Furion::n; i++)
    //{
    //    fileout << this->PHI[i] << " ";
    //}
    //fileout << "\n";
    //for (int i = 0; i < Furion::n; i++)
    //{
    //    fileout << this->PSI[i] << " ";
    //}
    //fileout << "\n";

    //fileout.close();
    //delete[] L, M, N;
    ////exit(0);
    
    beam_out = new G_Beam((this->X_), (this->Y_), (this->PHI), (this->PSI), beam_in->lambda);
    
    //delete[] L, M, N;
    //delete[] T, T1;
    //delete[] Nx, Ny, Nz;
    //delete[] hslope;
    //delete[] L2, M2, N2;
    //delete[] X3, Y3, Z3, L3, M3, N3;
}

void G_Oe::source_to_oe(real_t* X1, real_t* Y1, real_t* Z1, real_t* L1, real_t* M1, real_t* N1, real_t* X, real_t* Y, real_t ds, real_t* L, real_t* M, real_t* N)
{
    int n = Furion::n;
    real_t* OS = new real_t[9];
    real_t* OS_0 = new real_t[9];
    real_t* OS_1 = new real_t[9];
    f_rx.furion_rotx(theta, OS_0);
    f_rz.furion_rotz(chi, OS_1);

    real_t* Z = new real_t[1];
    Z[0] = -ds;

    matrixMulti_33(OS, OS_0, OS_1);
    G_Oe::matrixMulti_3n(X1, Y1, Z1, OS, X, Y, Z, 0, n);

    matrixMulti_3nn(L1, M1, N1, OS, L, M, N, 0, n);

    delete[] Z, OS, OS_0, OS_1;
    cout << " G_Oe��source_to_oe" << endl;
}

void G_Oe::matrixMulti_33(real_t* matrix, real_t* matrix1, real_t* matrix2)  //XYZ:1*3; LMN:1*n
{
    for (int i = 0; i < 3; i++)
    {
        matrix[i] = matrix1[0] * matrix2[i] + matrix1[1] * matrix2[i + 3] + matrix1[2] * matrix2[i + 6];
        matrix[i + 3] = matrix1[3] * matrix2[i] + matrix1[4] * matrix2[i + 3] + matrix1[5] * matrix2[i + 6];
        matrix[i + 6] = matrix1[6] * matrix2[i] + matrix1[7] * matrix2[i + 3] + matrix1[8] * matrix2[i + 6];
    }

    //__syncthreads();
}

__global__ void Furion_NS::matrixMulti_3n_cuda(real_t* L2, real_t* M2, real_t* N2, real_t* matrix, real_t* L, real_t* M, real_t* N, real_t dx, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        L2[i] = matrix[0] * L[i] + matrix[1] * M[i] + matrix[2] * N[0];
        M2[i] = matrix[3] * L[i] + matrix[4] * M[i] + matrix[5] * N[0];
        N2[i] = matrix[6] * L[i] + matrix[7] * M[i] + matrix[8] * N[0] + dx;
    }

    __syncthreads();
}

void G_Oe::matrixMulti_3n(real_t* L2, real_t* M2, real_t* N2, real_t* matrix, real_t* L, real_t* M, real_t* N, real_t dx, int n)  //XYZ:1*3; LMN:1*n
{
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    //real_t* d_L2, * d_M2, * d_N2, * d_L, * d_M, * d_N, * d_matrix;
    real_t* d_N, * d_matrix;
    //hipMalloc((void**)&d_L2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_M2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_N2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_L, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_M, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N, 1 * sizeof(real_t));
    hipMalloc((void**)&d_matrix, 9 * sizeof(real_t));

    //hipMemcpy(d_L, L, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_M, M, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, 1 * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, 9 * sizeof(real_t), hipMemcpyHostToDevice);

    Furion_NS::matrixMulti_3n_cuda << <blocksPerGrid, threadsPerBlock >> > (L2, M2, N2, d_matrix, L, M, d_N, dx, n);

    //hipMemcpy(L2, d_L2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(M2, d_M2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(N2, d_N2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    //hipFree(d_L2);
    //hipFree(d_M2);
    //hipFree(d_N2);
    //hipFree(d_L);
    //hipFree(d_M);
    hipFree(d_N);
    hipFree(d_matrix);
}

__global__ void Furion_NS::matrixMulti_3nn_cuda(real_t* L2, real_t* M2, real_t* N2, real_t* matrix, real_t* L, real_t* M, real_t* N, real_t dx, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        L2[i] = matrix[0] * L[i] + matrix[1] * M[i] + matrix[2] * N[i];
        M2[i] = matrix[3] * L[i] + matrix[4] * M[i] + matrix[5] * N[i];
        N2[i] = matrix[6] * L[i] + matrix[7] * M[i] + matrix[8] * N[i] + dx;
    }

    __syncthreads();
}

void G_Oe::matrixMulti_3nn(real_t* L2, real_t* M2, real_t* N2, real_t* matrix, real_t* L, real_t* M, real_t* N, real_t dx, int n)  //XYZ:1*3; LMN:1*n
{
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    //real_t* d_L2, * d_M2, * d_N2, * d_L, * d_M, * d_N, * d_matrix;
    real_t* d_matrix;
    /*hipMalloc((void**)&d_L2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_L, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N, Furion::n * sizeof(real_t));*/
    hipMalloc((void**)&d_matrix, 9 * sizeof(real_t));

    //hipMemcpy(d_L, L, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_M, M, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_N, N, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, 9 * sizeof(real_t), hipMemcpyHostToDevice);

    Furion_NS::matrixMulti_3nn_cuda << <blocksPerGrid, threadsPerBlock >> > (L2, M2, N2, d_matrix, L, M, N, dx, n);

    //hipMemcpy(L2, d_L2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(M2, d_M2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(N2, d_N2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    //hipFree(d_L2);
    //hipFree(d_M2);
    //hipFree(d_N2);
    //hipFree(d_L);
    //hipFree(d_M);
    //hipFree(d_N);
    hipFree(d_matrix);
}

void G_Oe::matrixMulti(real_t* L2, real_t* M2, real_t* N2, real_t* matrix, real_t* L, real_t* M, real_t* N, int n)  //XYZ:1*3; LMN:1*n
{
    for (int i = 0; i < n; i++)
    {
        L2[i] = matrix[0] * L[i] + matrix[1] * M[i] + matrix[2] * N[i];
        M2[i] = matrix[3] * L[i] + matrix[4] * M[i] + matrix[5] * N[i];
        N2[i] = matrix[6] * L[i] + matrix[7] * M[i] + matrix[8] * N[i];
    }

    //__syncthreads();
}

__global__ void Furion_NS::intersection_cuda(real_t* T, real_t* X2, real_t* Y2, real_t* Z2, real_t* X1, real_t* Y1, real_t* Z1, real_t* L1, real_t* M1, real_t* N1, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        T[i] = Y1[i] / M1[i];
        X2[i] = X1[i] + T[i] * L1[i];
        Y2[i] = 0;
        Z2[i] = Z1[i] + T[i] * N1[i];
    }

    __syncthreads();
}


void G_Oe::intersection(real_t* T, real_t* X2, real_t* Y2, real_t* Z2, real_t* X1, real_t* Y1, real_t* Z1, real_t* L1, real_t* M1, real_t* N1)
{
    int n = Furion::n;

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    /*real_t* d_X2, * d_Y2, * d_Z2;
    real_t* d_X1, * d_Y1, * d_Z1;
    real_t* d_L1, * d_M1, * d_N1, * d_T;
    hipMalloc((void**)&d_X2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Z2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_L1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_M1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_N1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_X1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Y1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_Z1, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_T, Furion::n * sizeof(real_t));

    hipMemcpy(d_L1, L1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_M1, M1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_N1, N1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_X1, X1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_Y1, Y1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_Z1, Z1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_X2, X2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_Y2, Y2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_Z2, Z2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);*/

    Furion_NS::intersection_cuda << <blocksPerGrid, threadsPerBlock >> > (T, X2, Y2, Z2, X1, Y1, Z1, L1, M1, N1, n);

    //hipMemcpy(T, d_T, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->X2, d_X2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->Y2, d_Y2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->Z2, d_Z2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    //hipFree(d_X1);
    //hipFree(d_Y1);
    //hipFree(d_Z1);
    //hipFree(d_X2);
    //hipFree(d_Y2);
    //hipFree(d_Z2);
    //hipFree(d_L1);
    //hipFree(d_M1);
    //hipFree(d_N1);
    //hipFree(d_T);

    cout << " G_Oe��intersection" << endl;

}

__global__ void Furion_NS::normal_cuda(real_t* Nx, real_t* Ny, real_t* Nz, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n)
    {
        Ny[i] = 0;
        Nz[i] = 0;
        Nx[i] = 0;
    }

    __syncthreads();
}


void G_Oe::normal(real_t* X2, real_t* Y2, real_t* Z2, real_t* Nx, real_t* Ny, real_t* Nz)
{
    int n = Furion::n;

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    Furion_NS::normal_cuda << <blocksPerGrid, threadsPerBlock >> > (Nx, Ny, Nz, n);

    cout << "G_Oe��normal" << endl;
}

__global__ void Furion_NS::h_slope1_cuda(real_t* delta_X, real_t* delta_Z, real_t* L1, real_t* N1, real_t* X2, real_t* Z2, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    real_t delta_Z1;

    if (i < n)
    {
        delta_Z1 = sqrt(L1[i] * L1[i] + N1[i] * N1[i]);
        delta_Z[i] = 1e-10 * N1[i] * delta_Z1 + Z2[i];
        delta_X[i] = 1e-10 * L1[i] * delta_Z1 + X2[i];
    }

    __syncthreads();
}

__global__ void Furion_NS::h_slope2_cuda(real_t* h_slope, real_t* Y2, real_t* h0, real_t* h1, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n)
    {
        h_slope[i] = (h1[i] - h0[i]) / 1e-10;
        Y2[i] = h0[i] + Y2[i];
    }

    __syncthreads();
}

void G_Oe::h_slope(real_t* X2, real_t* Y2, real_t* Z2, real_t* L1, real_t* N1, real_t* h_slope)
{
    int n = Furion::n;

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (Furion::n + threadsPerBlock - 1) / threadsPerBlock;

    //real_t* h0 = new real_t[Furion::n];
    //real_t* h1 = new real_t[Furion::n];
    //real_t* delta_Z = new real_t[Furion::n];
    //real_t* delta_X = new real_t[Furion::n];

    //real_t* d_L1, * d_N1, * d_X2, * d_Y2, * d_Z2;
    real_t* d_h0, * d_h1;
    real_t* d_delta_X, * d_delta_Z;

    //hipMalloc((void**)&d_L1, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_N1, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_X2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Y2, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_Z2, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_h0, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_h1, Furion::n * sizeof(real_t));
    //hipMalloc((void**)&d_h_slope, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_delta_X, Furion::n * sizeof(real_t));
    hipMalloc((void**)&d_delta_Z, Furion::n * sizeof(real_t));

    //hipMemcpy(d_L1, this->L1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_N1, this->N1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_X2, this->X2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_Y2, this->Y2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_Z2, this->Z2, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);

    surface->value(d_h0, Z2, X2, n);
    //hipMemcpy(d_h0, h0, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    //hipFree(d_h0);

    Furion_NS::h_slope1_cuda << <blocksPerGrid, threadsPerBlock >> > (d_delta_X, d_delta_Z, L1, N1, X2, Z2, n);
    //hipFree(d_L1);
    //hipFree(d_N1);
    //hipFree(d_X2);
    //hipFree(d_Z2);

    //hipMemcpy(delta_X, d_delta_X, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(delta_Z, d_delta_Z, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipFree(d_delta_X);
    //hipFree(d_delta_Z);

    surface->value(d_h1, d_delta_Z, d_delta_X, n);
    hipFree(d_delta_X);
    hipFree(d_delta_Z);
    //hipMemcpy(d_h1, h1, Furion::n * sizeof(real_t), hipMemcpyHostToDevice);
    Furion_NS::h_slope2_cuda << <blocksPerGrid, threadsPerBlock >> > (h_slope, Y2, d_h0, d_h1, n);
    //hipMemcpy(h_slope, d_h_slope, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);
    //hipMemcpy(this->Y2, d_Y2, Furion::n * sizeof(real_t), hipMemcpyDeviceToHost);

    hipFree(d_h0);
    hipFree(d_h1);

    //hipFree(d_Y2);
    //hipFree(d_h_slope);

    //delete[] h0, h1, delta_Z, delta_X;
    cout << " G_Oe��h_slope" << endl;
}

void G_Oe::oe_to_image(real_t* X3, real_t* Y3, real_t* Z3, real_t* L3, real_t* M3, real_t* N3, real_t* X2, real_t* Y2, real_t* Z2, real_t di, real_t* L2, real_t* M2, real_t* N2)
{
    int n = Furion::n;

    real_t* OS = new real_t[9];
    real_t* OS_0 = new real_t[9];
    real_t* OS_1 = new real_t[9];

    real_t* X0 = new real_t[Furion::n];
    real_t* Y0 = new real_t[Furion::n];
    real_t* Z0 = new real_t[Furion::n];

    f_rx.furion_rotx(this->theta2, OS_0);
    G_Oe::matrixMulti_3nn(X3, Y3, Z3, OS_0, X2, Y2, Z2, -di, n);

    f_rz.furion_rotz(-1 * this->chi, OS_0);
    f_rx.furion_rotx(this->theta2, OS_1);
    G_Oe::matrixMulti_33(OS, OS_0, OS_1);
    G_Oe::matrixMulti_3nn(L3, M3, N3, OS, L2, M2, N2, 0, n);

    delete[] X0, Y0, Z0, OS, OS_0, OS_1;
    cout << " G_Oe��oe_to_image" << endl;
}
