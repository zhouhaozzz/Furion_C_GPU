#include "hip/hip_runtime.h"
#include "Furion.h"
#include "grating.h"
#include "g_source.h"
#include "g_beam.h"
#include "g_Furion_cylinder_ellipse_Mirror.h"
//#include "g_Furion_ellipsoid_Mirror.h"
#include "no_surfe.h"
#include <chrono>
#define Pi 3.1415926536
#define E  2.71828

using namespace Furion_NS;

Furion::Furion(int rank1, int size1)
{
	std::cout << std::fixed;
	std::cout << std::setprecision(15);

#ifdef CUDA
	cout << "Furion" << endl;
	real_t wavelength = 1e-8;
	int n = Furion::n;
	grating = new Grating(230e3, 2.0984e-2, 0, wavelength);
	no_surfe = new No_Surfe();

	for (int i = 0; i < sizeof(pre_Mirror_theta) / sizeof(real_t); i++)
	{
		Lambda[i] = Lambda[i] * 1e-9;
		pre_Mirror_theta[i] = pre_Mirror_theta[i] / 180 * Pi;
		grating_theta[i] = grating_theta[i] * 1e-3;
		beamsize[i] = beamsize[i] * 1e-3 / (2 * sqrt(2 * log(2)));
		divergence[i] = divergence[i] * 1e-6 / (2 * sqrt(2 * log(2)));
	}
	lambda = Lambda[i];
	psigmax = beamsize[i];
	psigmay = beamsize[i];
	vsigmax = divergence[i];
	vsigmay = divergence[i];//

	g_source = new G_Source(psigmax, vsigmax, Furion::n, lambda, rank1);
	G_Beam b1 = g_source->beam_out.translate(196);

	auto start = std::chrono::high_resolution_clock::now();
	srand((unsigned)time(NULL));

	g_Furion_cylinder_ellipse_Mirror = new G_Furion_Cylinder_Ellipse_Mirror(&b1, 0, 0, 0, 7e-3, no_surfe, 196, 98, grating);
	g_Furion_cylinder_ellipse_Mirror->run(&b1, 0, 0, 0, 7e-3, no_surfe, 196, 98, grating);
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "Execution time: " << duration.count() / 1e6 << " seconds" << std::endl;
	if (0)
	{
		G_Beam* b2 = g_Furion_cylinder_ellipse_Mirror->beam_out; b2->plot_sigma(0, rank1);
		std::string inputString = std::to_string(size1);
		std::string command = ("python python_plot/Furion_plot4_6sigma.py " + inputString);
		int returnCode = system(command.c_str());
		if (returnCode != 0)
		{
			std::cerr << "Python drawing script execution failed!" << std::endl;
		}
	}
	


#endif 

}

Furion::~Furion()
{

	//delete g_Furion_cylinder_ellipse_Mirror;

}

// real_t sum = 0;
// real_t sum1 = 0;
//     for (int i = 0; i < Furion::n; i++)
//     {
//         sum = sum + fabs(this->Phase[i] );
//         sum1 = sum1 + fabs(this->X_[i] );
//     }
//         cout << sum << " " << sum1 <<endl;